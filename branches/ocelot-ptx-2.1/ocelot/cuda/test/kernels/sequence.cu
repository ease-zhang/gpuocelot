/*!
	\brief sequence.cu
	\author Andrew Kerr

	\brief simple test of a CUDA implementation's ability to allocate memory on the device, launch
		a kernel, and fetch its results. One kernel requires no syncthreads, another kernel requires
		one synchronization
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" __global__ void sequence(int *A, int N) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < N) {
		A[i] = 2*i;
	}
}

extern "C" __global__ void testShr(int *A, const int *B) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int b;
	__shared__ int storage[256];
	
	storage[threadIdx.x] = B[i];
	__syncthreads();
	if (i & 1) {
		b = storage[threadIdx.x ^ 1] * 2;
	}
	else {
		b = storage[threadIdx.x ^ 1] * 3;
	}
	A[i] = b;
}

extern "C" __global__ void v4sequence(int4 *A, int N) {
	int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int4 b = make_int4(i, 2*i, 3*i, 4*i);
	A[i-1] = b;
}

int main(int argc, char *arg[]) {

	const int N = 1024;
	int *A_host, *A_gpu =0;
	int errors = 0;

	size_t bytes = sizeof(int)*N;

	if (hipMalloc((void **)&A_gpu, bytes) != hipSuccess) {
		printf("cudaMalloc() - failed to allocate %d bytes on device\n", (int)bytes);
		return -1;
	}

	A_host = (int *)malloc(bytes);
	for (int i = 0; i < N; i++) {
		A_host[i] = -1;
	}
	
	hipMemcpy(A_gpu, A_host, bytes, hipMemcpyHostToDevice);
	
	printf("A_host = 0x%x\n", (void *)A_host);
	printf("A_gpu = 0x%x\n", (void *)A_gpu);

	dim3 grid((N+31)/32,1);
	dim3 block(32, 1);
	
	sequence<<< grid, block >>>(A_gpu, N);
	
	printf("cudaMemcpy(0x%x, 0x%x) - APP\n", (void *)A_host, (void *)A_gpu);
	hipMemcpy(A_host, A_gpu, bytes, hipMemcpyDeviceToHost);
	for (int i = 0; i < N && errors < 5; i++) {
		if (A_host[i] != 2*i) {
			
			printf("ERROR 1 [%d] - expected: %d, got: %d\n", i, 2*i, A_host[i]);
			++errors;
		}
	}
	
	grid.x /= 4;
	v4sequence<<< grid, block >>>((int4 *)A_gpu, N/4);
	hipMemcpy(A_host, A_gpu, bytes, hipMemcpyDeviceToHost);
	grid.x *= 4;

	int *B_gpu = 0;
	if (hipMalloc((void **)&B_gpu, bytes) != hipSuccess) {
		printf("cudaMalloc() - failed to allocate %d bytes on device\n", (int)bytes);
		hipFree(A_gpu);
		free(A_host);
		return -1;
	}
	
	sequence<<< grid, block >>>(A_gpu, N);
	testShr<<< grid, block >>>(B_gpu, A_gpu);
	
	if (hipMemcpy(A_host, B_gpu, bytes, hipMemcpyDeviceToHost) != hipSuccess) {
		printf("cudaMemcpy(A, B) - failed to copy %d bytes from device to host\n", (int)bytes);
		hipFree(A_gpu);
		hipFree(B_gpu);
		free(A_host);
	}
	
	for (int i = 0; (errors < 5) && i < N; ++i) {
		int b;
		if (i & 1) {
			b = (i ^ 1) * 2 * 2;
		}
		else {
			b = (i ^ 1) * 2 * 3;
		}
		int got = A_host[i];
		if (b != got) {
			printf("ERROR 2 [%d] - expected: %d, got: %d\n", i, b, got);
			++errors;
		}
	}

	hipFree(B_gpu);
	hipFree(A_gpu);
	free(A_host);


	if (errors) {
		printf("Pass/Fail : Fail\n");
	}
	else {
		printf("Pass/Fail : Pass\n");
	}

	return 0;
}
